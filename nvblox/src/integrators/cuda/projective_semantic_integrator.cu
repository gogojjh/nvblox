#include "hip/hip_runtime.h"
/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include <nvblox/integrators/projective_semantic_integrator.h>

#include "nvblox/core/color.h"
// #include "nvblox/core/color_map.h"
#include "nvblox/core/cuda/error_check.cuh"
#include "nvblox/core/interpolation_2d.h"
#include "nvblox/integrators/internal/cuda/projective_integrators_common.cuh"
#include "nvblox/integrators/internal/integrators_common.h"
#include "nvblox/utils/semantickitti_label.h"
#include "nvblox/utils/timing.h"
#include "nvblox/utils/weight_function.h"

namespace nvblox {
template void ProjectiveSemanticIntegrator::integrateCameraFrame(
    const SemanticImage& semantic_frame, const Transform& T_L_C,
    const Camera& camera, const TsdfLayer& tsdf_layer,
    SemanticLayer* semantic_layer, std::vector<Index3D>* updated_blocks);

template void ProjectiveSemanticIntegrator::integrateCameraFrame(
    const SemanticImage& semantic_frame, const Transform& T_L_C,
    const CameraPinhole& camera, const TsdfLayer& tsdf_layer,
    SemanticLayer* semantic_layer, std::vector<Index3D>* updated_blocks);
}  // namespace nvblox

namespace nvblox {
__device__ inline bool updateSemanticVoxel(const uint16_t semantic_label,
                                           SemanticVoxel* voxel_ptr) {
  uint16_t update_label;
  nvblox::semantic_kitti::normalizeSemanticKittiLabel(semantic_label,
                                                      &update_label);

  // updateSemanticVoxelProbabilities
  SemanticProbabilities semantic_label_frequencies;
  semantic_label_frequencies.setZero();
  if (update_label >= semantic_label_frequencies.size()) {
    return false;
  }
  semantic_label_frequencies[update_label] += 1.0f;

  const float log_match_probability_ = logf(0.8f);
  const float log_non_match_probability_ = logf(0.2f);

  // TODO(gogojjh):
  // A `#Labels X #Labels` Eigen matrix where each `j` column represents the
  // probability of observing label `j` when current label is `i`, where `i`
  // is the row index of the matrix.
  // Eigen::Matrix<float, kTotalNumberOfLabels, kTotalNumberOfLabels>
  //     semantic_log_likelihood_;
  // semantic_log_likelihood_ =
  //     semantic_log_likelihood_.Constant(log_non_match_probability_);
  // semantic_log_likelihood_.diagonal() =
  //     semantic_log_likelihood_.diagonal().Constant(log_match_probability_);

  voxel_ptr->semantic_priors +=
      log_match_probability_ * semantic_label_frequencies;

  // updateSemanticVoxel label by the MLE
  voxel_ptr->semantic_priors.maxCoeff(&voxel_ptr->semantic_label);
  return true;
}

__device__ inline bool interpolateLidarImage(
    const Lidar& lidar, const Vector3f& p_voxel_center_C, const float* image,
    const Vector2f& u_px, const int rows, const int cols,
    const float linear_interpolation_max_allowable_difference_m,
    const float nearest_interpolation_max_allowable_squared_dist_to_ray_m,
    float* image_value) {
  // Try linear interpolation first
  interpolation::Interpolation2DNeighbours<float> neighbours;
  bool linear_interpolation_success = interpolation::interpolate2DLinear<
      float, interpolation::checkers::FloatPixelGreaterThanZero>(
      image, u_px, rows, cols, image_value, &neighbours);

  // Additional check
  // Check that we're not interpolating over a discontinuity
  // NOTE(alexmillane): This prevents smearing are object edges.
  if (linear_interpolation_success) {
    const float d00 = fabsf(neighbours.p00 - *image_value);
    const float d01 = fabsf(neighbours.p01 - *image_value);
    const float d10 = fabsf(neighbours.p10 - *image_value);
    const float d11 = fabsf(neighbours.p11 - *image_value);
    float maximum_depth_difference_to_neighbours =
        fmax(fmax(d00, d01), fmax(d10, d11));
    if (maximum_depth_difference_to_neighbours >
        linear_interpolation_max_allowable_difference_m) {
      linear_interpolation_success = false;
    }
  }

  // If linear didn't work - try nearest neighbour interpolation
  if (!linear_interpolation_success) {
    Index2D u_neighbour_px;
    if (!interpolation::interpolate2DClosest<
            float, interpolation::checkers::FloatPixelGreaterThanZero>(
            image, u_px, rows, cols, image_value, &u_neighbour_px)) {
      // If we can't successfully do closest, fail to intgrate this voxel.
      return false;
    }
    // Additional check
    // Check that this voxel is close to the ray passing through the pixel.
    // Note(alexmillane): This is to prevent large numbers of voxels
    // being integrated by a single pixel at long ranges.
    const Vector3f closest_ray = lidar.vectorFromPixelIndices(u_neighbour_px);
    const float off_ray_squared_distance =
        (p_voxel_center_C - p_voxel_center_C.dot(closest_ray) * closest_ray)
            .squaredNorm();
    if (off_ray_squared_distance >
        nearest_interpolation_max_allowable_squared_dist_to_ray_m) {
      return false;
    }
  }

  // TODO(alexmillane): We should add clearing rays, even in the case both
  // interpolations fail.
  return true;
}

// nearest_interpolation_max_allowable_squared_dist_to_ray_m, default: 0.125 **2
__device__ inline bool interpolateOSLidarImage(
    const OSLidar& lidar, const Vector3f& p_voxel_center_C, const float* image,
    const Vector2f& u_px, const int rows, const int cols,
    const float linear_interpolation_max_allowable_difference_m,
    const float nearest_interpolation_max_allowable_squared_dist_to_ray_m,
    float* image_value) {
  // Try linear interpolation first
  interpolation::Interpolation2DNeighbours<float> neighbours;
  bool linear_interpolation_success = interpolation::interpolate2DLinear<
      float, interpolation::checkers::FloatPixelGreaterThanZero>(
      image, u_px, rows, cols, image_value, &neighbours);

  // Additional check
  // Check that we're not interpolating over a discontinuity
  // NOTE(alexmillane): This prevents smearing are object edges.
  if (linear_interpolation_success) {
    const float d00 = fabsf(neighbours.p00 - *image_value);
    const float d01 = fabsf(neighbours.p01 - *image_value);
    const float d10 = fabsf(neighbours.p10 - *image_value);
    const float d11 = fabsf(neighbours.p11 - *image_value);
    float maximum_depth_difference_to_neighbours =
        fmax(fmax(d00, d01), fmax(d10, d11));
    if (maximum_depth_difference_to_neighbours >
        linear_interpolation_max_allowable_difference_m) {
      linear_interpolation_success = false;
    }
  }

  // If linear didn't work - try nearest neighbour interpolation
  if (!linear_interpolation_success) {
    Index2D u_neighbour_px;
    if (!interpolation::interpolate2DClosest<
            float, interpolation::checkers::FloatPixelGreaterThanZero>(
            image, u_px, rows, cols, image_value, &u_neighbour_px)) {
      // If we can't successfully do closest, fail to intgrate this voxel.
      return false;
    }
    // Additional check
    // Check that this voxel is close to the ray passing through the pixel.
    // Note(alexmillane): This is to prevent large numbers of voxels
    // being integrated by a single pixel at long ranges.
    const Vector3f closest_ray = lidar.vectorFromPixelIndices(u_neighbour_px);
    const float off_ray_squared_distance =
        (p_voxel_center_C - p_voxel_center_C.dot(closest_ray) * closest_ray)
            .squaredNorm();
    if (off_ray_squared_distance >
        nearest_interpolation_max_allowable_squared_dist_to_ray_m) {
      return false;
    }
  }

  // TODO(alexmillane): We should add clearing rays, even in the case both
  // interpolations fail.
  return true;
}

__device__ inline bool getPointVectorOSLidar(const OSLidar& lidar,
                                             const Index2D& u_C, const int rows,
                                             const int cols,
                                             Vector3f& point_vector) {
  const float kFloatEpsilon = 1e-8;  // Used for weights
  if (u_C.x() < 0 || u_C.y() < 0 || u_C.x() >= cols || u_C.y() >= rows) {
    return false;
  } else {
    point_vector = lidar.unprojectFromImageIndex(u_C);
    if (point_vector.norm() < kFloatEpsilon) {
      return false;
    } else {
      return true;
    }
  }
}

__device__ inline bool getNormalVectorOSLidar(const OSLidar& lidar,
                                              const Index2D& u_C,
                                              const int rows, const int cols,
                                              Vector3f& normal_vector) {
  const float kFloatEpsilon = 1e-8;  // Used for weights
  if (u_C.x() < 0 || u_C.y() < 0 || u_C.x() >= cols || u_C.y() >= rows) {
    return false;
  } else {
    normal_vector = lidar.getNormalVector(u_C);
    if (normal_vector.norm() < kFloatEpsilon) {
      return false;
    } else {
      return true;
    }
  }
}

// CAMERA
// __global__ void integrateBlocksKernel(const Index3D*
// block_indices_device_ptr,
//                                       const Camera camera, const float*
//                                       image, int rows, int cols, const
//                                       Transform T_C_L, const float
//                                       block_size, const float
//                                       truncation_distance_m, const float
//                                       max_weight, const float
//                                       max_integration_distance, TsdfBlock**
//                                       block_device_ptrs) {
//   // Get - the image-space projection of the voxel associated with this
//   // thread
//   //     - the depth associated with the projection.
//   Eigen::Vector2f u_px;
//   float voxel_depth_m;
//   Vector3f p_voxel_center_C;
//   if (!projectThreadVoxel(block_indices_device_ptr, camera, T_C_L,
//   block_size,
//                           &u_px, &voxel_depth_m, &p_voxel_center_C)) {
//     return;
//   }

//   // If voxel further away than the limit, skip this voxel
//   if (max_integration_distance > 0.0f) {
//     if (voxel_depth_m > max_integration_distance) {
//       return;
//     }
//   }

//   // Interpolate on the image plane
//   float image_value;
//   if (!interpolation::interpolate2DClosest<
//           float, interpolation::checkers::FloatPixelGreaterThanZero>(
//           image, u_px, rows, cols, &image_value)) {
//     return;
//   }

//   // Get the Voxel we'll update in this thread
//   // NOTE(alexmillane): Note that we've reverse the voxel indexing order
//   // such that adjacent threads (x-major) access adjacent memory locations
//   // in the block (z-major).
//   TsdfVoxel* voxel_ptr = &(block_device_ptrs[blockIdx.x]
//                                ->voxels[threadIdx.z][threadIdx.y][threadIdx.x]);

//   // Update the voxel using the update rule for this layer type
//   updateVoxel(image_value, voxel_ptr, voxel_depth_m, truncation_distance_m,
//               max_weight);
// }

// OSLiDAR
__global__ void integrateBlocksKernel(
    const Index3D* block_indices_device_ptr, const OSLidar lidar,
    const uint16_t* semantic_image, const float* depth_image, int rows,
    int cols, const Transform T_C_L, const float block_size,
    const float truncation_distance_m, const float max_weight,
    const float max_integration_distance,
    const float linear_interpolation_max_allowable_difference_m,
    const float nearest_interpolation_max_allowable_squared_dist_to_ray_m,
    SemanticBlock** block_device_ptrs) {
  // function 1
  // Get - the image-space projection of the voxel associated with this
  // thread
  //     - the depth associated with the projection.
  //     - the projected image coordinate of the voxel
  Eigen::Vector2f u_px;
  float voxel_depth_m;
  Vector3f p_voxel_center_C;
  if (!projectThreadVoxel(block_indices_device_ptr, lidar, T_C_L, block_size,
                          &u_px, &voxel_depth_m, &p_voxel_center_C)) {
    return;  // false: the voxel is not visible
  }

  // If voxel further away than the limit, skip this voxel
  if (max_integration_distance > 0.0f) {
    if (voxel_depth_m > max_integration_distance) {
      return;
    }
  }

  // function 2: Interpolate on the depth_image plane
  float depth_image_value;
  if (!interpolateOSLidarImage(
          lidar, p_voxel_center_C, depth_image, u_px, rows, cols,
          linear_interpolation_max_allowable_difference_m,
          nearest_interpolation_max_allowable_squared_dist_to_ray_m,
          &depth_image_value)) {
    return;
  }

  // function 3: Occlusion testing
  // Get the distance of the voxel from the rendered surface. If outside
  // truncation band, skip.
  const float voxel_distance_from_surface = depth_image_value - voxel_depth_m;
  if (fabsf(voxel_distance_from_surface) > truncation_distance_m) {
    return;
  }

  // function 4: Get the closest semantic value
  // If we can't successfully do closest, fail to intgrate this voxel.
  uint16_t semantic_image_value;
  if (!interpolation::interpolate2DClosest<
          uint16_t, interpolation::checkers::PixelAlwaysValid<uint16_t>>(
          semantic_image, u_px, rows, cols, &semantic_image_value)) {
    return;
  }

  // Get the Voxel we'll update in this thread
  // NOTE(alexmillane): Note that we've reverse the voxel indexing order
  // such that adjacent threads (x-major) access adjacent memory locations
  // in the block (z-major).
  SemanticVoxel* voxel_ptr =
      &(block_device_ptrs[blockIdx.x]
            ->voxels[threadIdx.z][threadIdx.y][threadIdx.x]);

  // Update the semantic voxel
  updateSemanticVoxel(semantic_image_value, voxel_ptr);
}

__global__ void updateColorBlocks(
    const SemanticBlock** block_device_ptrs_semantic,
    ColorBlock** block_device_ptrs_color) {
  const SemanticVoxel* semantic_voxel_ptr =
      &(block_device_ptrs_semantic[blockIdx.x]
            ->voxels[threadIdx.z][threadIdx.y][threadIdx.x]);

  ColorVoxel* color_voxel_ptr =
      &(block_device_ptrs_color[blockIdx.x]
            ->voxels[threadIdx.z][threadIdx.y][threadIdx.x]);

  Index3D color;  // bgr
  nvblox::semantic_kitti::updateLabelColorMap(
      semantic_voxel_ptr->semantic_label, &color);
  color_voxel_ptr->color = Color(color.z(), color.y(), color.x());
}

ProjectiveSemanticIntegrator::ProjectiveSemanticIntegrator()
    : ProjectiveIntegratorBase() {
  checkCudaErrors(hipStreamCreate(&integration_stream_));
}

ProjectiveSemanticIntegrator::~ProjectiveSemanticIntegrator() {
  finish();
  checkCudaErrors(hipStreamDestroy(integration_stream_));
}

void ProjectiveSemanticIntegrator::finish() const {
  hipStreamSynchronize(integration_stream_);
}

float ProjectiveSemanticIntegrator::
    lidar_linear_interpolation_max_allowable_difference_vox() const {
  return lidar_linear_interpolation_max_allowable_difference_vox_;
}

float ProjectiveSemanticIntegrator::
    lidar_nearest_interpolation_max_allowable_dist_to_ray_vox() const {
  return lidar_nearest_interpolation_max_allowable_dist_to_ray_vox_;
}

void ProjectiveSemanticIntegrator::
    lidar_linear_interpolation_max_allowable_difference_vox(float value) {
  CHECK_GT(value, 0.0f);
  lidar_linear_interpolation_max_allowable_difference_vox_ = value;
}

void ProjectiveSemanticIntegrator::
    lidar_nearest_interpolation_max_allowable_dist_to_ray_vox(float value) {
  CHECK_GT(value, 0.0f);
  lidar_nearest_interpolation_max_allowable_dist_to_ray_vox_ = value;
}

template <typename CameraType>
void ProjectiveSemanticIntegrator::integrateCameraFrame(
    const SemanticImage& semantic_frame, const Transform& T_L_C,
    const CameraType& camera, const TsdfLayer& tsdf_layer,
    SemanticLayer* semantic_layer, std::vector<Index3D>* updated_blocks) {
  LOG(INFO) << "ProjectiveSemanticIntegrator::integrateFrame<CameraType>";
  CHECK_NOTNULL(semantic_layer);
  // updateCameraBlocks(block_indices, depth_frame, semantic_frame, T_L_C,
  // lidar,
  //                   semantic_layer);
}

void ProjectiveSemanticIntegrator::integrateLidarFrame(
    const DepthImage& depth_frame, const SemanticImage& semantic_frame,
    const Transform& T_L_C, const OSLidar& lidar, const TsdfLayer& tsdf_layer,
    SemanticLayer* semantic_layer, std::vector<Index3D>* updated_blocks) {
  CHECK_NOTNULL(semantic_layer);
  timing::Timer tsdf_timer("semantic/integrate");

  // Metric truncation distance for this layer
  const float voxel_size =
      semantic_layer->block_size() / VoxelBlock<bool>::kVoxelsPerSide;
  const float truncation_distance_m = truncation_distance_vox_ * voxel_size;
  LOG(INFO) << "[semantic] Truncation distance: " << truncation_distance_m;

  // Identify blocks we can (potentially) see
  timing::Timer blocks_in_view_timer("semantic/integrate/get_blocks_in_view");
  std::vector<Index3D> block_indices =
      view_calculator_.getBlocksInImageViewRaycast(
          depth_frame, T_L_C, lidar, semantic_layer->block_size(),
          truncation_distance_m, max_integration_distance_m_);
  blocks_in_view_timer.Stop();

  // ***********************************************************
  // NOTE(gogojjh): need to check the function
  // Check which of these blocks are:
  // - Allocated in the TSDF, and
  // - have at least a single voxel within the truncation band
  // This is because:
  // - We don't allocate new geometry here, we just color existing geometry
  // - We don't color freespace.
  timing::Timer blocks_in_band_timer(
      "semantic/integrate/reduce_to_blocks_in_band");
  block_indices = reduceBlocksToThoseInTruncationBand(block_indices, tsdf_layer,
                                                      truncation_distance_m);
  // NOTE(gogojjh): comment to be removed
  // LOG(INFO) << "[semantic] (remining after removal) block_indices size: "
  //           << block_indices.size();
  blocks_in_band_timer.Stop();
  // ***********************************************************

  // Allocate blocks (CPU)
  timing::Timer allocate_blocks_timer("semantic/integrate/allocate_blocks");
  allocateBlocksWhereRequired(block_indices, semantic_layer);
  allocate_blocks_timer.Stop();

  // Update identified blocks
  timing::Timer update_blocks_timer("semantic/integrate/update_blocks");
  integrateBlocksTemplate(block_indices, depth_frame, semantic_frame, T_L_C,
                          lidar, semantic_layer);
  update_blocks_timer.Stop();

  if (updated_blocks != nullptr) {
    *updated_blocks = block_indices;
  }
}

template <typename SensorType>
void ProjectiveSemanticIntegrator::integrateBlocksTemplate(
    const std::vector<Index3D>& block_indices, const DepthImage& depth_frame,
    const SemanticImage& semantic_frame, const Transform& T_L_C,
    const SensorType& sensor, SemanticLayer* layer_ptr) {
  CHECK_NOTNULL(layer_ptr);

  if (block_indices.empty()) {
    return;
  }
  const int num_blocks = block_indices.size();

  // Expand the buffers when needed
  if (num_blocks > block_indices_device_.size()) {
    constexpr float kBufferExpansionFactor = 1.5f;
    const int new_size = static_cast<int>(kBufferExpansionFactor * num_blocks);
    block_indices_device_.reserve(new_size);
    block_ptrs_device_.reserve(new_size);
    block_indices_host_.reserve(new_size);
    block_ptrs_host_.reserve(new_size);
  }

  // Stage on the host pinned memory
  block_indices_host_ = block_indices;
  block_ptrs_host_ = getBlockPtrsFromIndices(block_indices, layer_ptr);

  // Transfer to the device
  block_indices_device_ = block_indices_host_;
  block_ptrs_device_ = block_ptrs_host_;

  // We need the inverse transform in the kernel
  const Transform T_C_L = T_L_C.inverse();

  integrateBlocks(depth_frame, semantic_frame, T_C_L, sensor, layer_ptr);
}

// OSLidar
void ProjectiveSemanticIntegrator::integrateBlocks(
    const DepthImage& depth_frame, const SemanticImage& semantic_frame,
    const Transform& T_C_L, const OSLidar& lidar, SemanticLayer* layer_ptr) {
  LOG(INFO) << "updateBLocks";
  /// block integration
  // Kernel call - One ThreadBlock launched per VoxelBlock
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const dim3 kThreadsPerBlock(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  // NOTE(gogojjh): the number of visible blocks
  const int num_thread_blocks = block_indices_device_.size();

  // Metric truncation distance for this layer
  const float voxel_size =
      layer_ptr->block_size() / VoxelBlock<bool>::kVoxelsPerSide;
  // default: 4.0 * 0.1
  const float truncation_distance_m = truncation_distance_vox_ * voxel_size;

  // Metric params
  const float linear_interpolation_max_allowable_difference_m =
      lidar_linear_interpolation_max_allowable_difference_vox_ * voxel_size;
  const float nearest_interpolation_max_allowable_squared_dist_to_ray_m =
      std::pow(lidar_nearest_interpolation_max_allowable_dist_to_ray_vox_ *
                   voxel_size,
               2);

  integrateBlocksKernel<<<num_thread_blocks, kThreadsPerBlock, 0,
                          integration_stream_>>>(
      block_indices_device_.data(),                               // NOLINT
      lidar,                                                      // NOLINT
      semantic_frame.dataConstPtr(),                              // NOLINT
      depth_frame.dataConstPtr(),                                 // NOLINT
      depth_frame.rows(),                                         // NOLINT
      depth_frame.cols(),                                         // NOLINT
      T_C_L,                                                      // NOLINT
      layer_ptr->block_size(),                                    // NOLINT
      truncation_distance_m,                                      // NOLINT
      max_weight_,                                                // NOLINT
      max_integration_distance_m_,                                // NOLINT
      linear_interpolation_max_allowable_difference_m,            // NOLINT
      nearest_interpolation_max_allowable_squared_dist_to_ray_m,  // NOLINT
      block_ptrs_device_.data());                                 // NOLINT

  // Finish processing of the frame before returning control
  finish();
  checkCudaErrors(hipPeekAtLastError());
}

// NOTE(gogojjh): synchronize color with semantic
void ProjectiveSemanticIntegrator::updateColorLayer(
    const std::vector<Index3D>& block_indices,
    const SemanticLayer& semantic_layer, ColorLayer* layer_ptr) {
  CHECK_NOTNULL(layer_ptr);

  if (block_indices.empty()) {
    return;
  }
  const int num_blocks = block_indices.size();
  allocateBlocksWhereRequired(block_indices, layer_ptr);

  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const dim3 kThreadsPerBlock(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);

  // Stage on the host pinned memory
  block_indices_host_ = block_indices;

  host_vector<const SemanticBlock*> block_ptrs_host_semantic;
  block_ptrs_host_semantic.reserve(num_blocks);
  device_vector<const SemanticBlock*> block_ptrs_device_semantic;
  block_ptrs_device_semantic.reserve(num_blocks);
  host_vector<ColorBlock*> block_ptrs_host_color;
  block_ptrs_host_color.reserve(num_blocks);
  device_vector<ColorBlock*> block_ptrs_device_color;
  block_ptrs_device_color.reserve(num_blocks);

  block_ptrs_host_semantic =
      getBlockPtrsFromIndices(block_indices, semantic_layer);
  block_ptrs_host_color = getBlockPtrsFromIndices(block_indices, layer_ptr);

  // Transfer to the device
  block_ptrs_device_semantic = block_ptrs_host_semantic;
  block_ptrs_device_color = block_ptrs_host_color;

  updateColorBlocks<<<num_blocks, kThreadsPerBlock, 0,
                      integration_stream_>>>(
      block_ptrs_device_semantic.data(),  // NOLINT
      block_ptrs_device_color.data());

  // Finish processing of the frame before returning control
  finish();
  checkCudaErrors(hipPeekAtLastError());
}

// *********************************************
// *********************************************
__global__ void checkBlocksInTruncationBandSemantics(
    const VoxelBlock<TsdfVoxel>** block_device_ptrs,
    const float truncation_distance_m,
    bool* contains_truncation_band_device_ptr) {
  // A single thread in each block initializes the output to 0
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    contains_truncation_band_device_ptr[blockIdx.x] = 0;
  }
  // An function of CUDA to synchronize threads
  __syncthreads();

  // Get the Voxel we'll check in this thread
  const TsdfVoxel voxel = block_device_ptrs[blockIdx.x]
                              ->voxels[threadIdx.z][threadIdx.y][threadIdx.x];

  // If this voxel in the truncation band, write the flag to say that the block
  // should be processed.
  // NOTE(alexmillane): There will be collision on write here. However, from my
  // reading, all threads' writes will result in a single write to global
  // memory. Because we only write a single value (1) it doesn't matter which
  // thread "wins".
  if (std::abs(voxel.distance) <= truncation_distance_m) {
    contains_truncation_band_device_ptr[blockIdx.x] = true;
  }
}

std::vector<Index3D>
ProjectiveSemanticIntegrator::reduceBlocksToThoseInTruncationBand(
    const std::vector<Index3D>& block_indices, const TsdfLayer& tsdf_layer,
    const float truncation_distance_m) {
  // Check 1) Are the blocks allocated
  // - performed on the CPU because the hash-map is on the CPU
  std::vector<Index3D> block_indices_check_1;
  block_indices_check_1.reserve(block_indices.size());
  for (const Index3D& block_idx : block_indices) {
    if (tsdf_layer.isBlockAllocated(block_idx)) {
      block_indices_check_1.push_back(block_idx);
    }
  }

  if (block_indices_check_1.empty()) {
    return block_indices_check_1;
  }

  // Check 2) Does each of the blocks have a voxel within the truncation band
  // - performed on the GPU because the blocks are there
  // Get the blocks we need to check
  std::vector<const TsdfBlock*> block_ptrs =
      getBlockPtrsFromIndices(block_indices_check_1, tsdf_layer);

  const int num_blocks = block_ptrs.size();
  LOG(INFO) << "num_blocks: " << num_blocks;

  // Expand the buffers when needed
  if (num_blocks > truncation_band_block_ptrs_device_.size()) {
    constexpr float kBufferExpansionFactor = 1.5f;
    const int new_size = static_cast<int>(kBufferExpansionFactor * num_blocks);
    truncation_band_block_ptrs_host_.reserve(new_size);
    truncation_band_block_ptrs_device_.reserve(new_size);
    block_in_truncation_band_device_.reserve(new_size);
    block_in_truncation_band_host_.reserve(new_size);
  }

  // Host -> Device
  truncation_band_block_ptrs_host_ = block_ptrs;
  truncation_band_block_ptrs_device_ = truncation_band_block_ptrs_host_;

  // Prepare output space
  block_in_truncation_band_device_.resize(num_blocks);

  // Do the check on GPU
  // Kernel call - One ThreadBlock launched per VoxelBlock
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const dim3 kThreadsPerBlock(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  const int num_thread_blocks = num_blocks;

  checkBlocksInTruncationBandSemantics<<<num_thread_blocks, kThreadsPerBlock, 0,
                                         integration_stream_>>>(
      truncation_band_block_ptrs_device_.data(), truncation_distance_m,
      block_in_truncation_band_device_.data());

  checkCudaErrors(hipStreamSynchronize(integration_stream_));
  checkCudaErrors(hipPeekAtLastError());

  // Copy results back
  block_in_truncation_band_host_ = block_in_truncation_band_device_;

  // Filter the indices using the result
  std::vector<Index3D> block_indices_check_2;
  block_indices_check_2.reserve(block_indices_check_1.size());
  for (int i = 0; i < block_indices_check_1.size(); i++) {
    if (block_in_truncation_band_host_[i] == true) {
      block_indices_check_2.push_back(block_indices_check_1[i]);
    }
  }

  return block_indices_check_2;
}

}  // namespace nvblox
