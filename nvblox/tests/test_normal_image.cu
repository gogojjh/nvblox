#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define STB_IMAGE_IMPLEMENTATION
#include "../executables/include/nvblox/datasets/external/stb_image.h"

#include <iostream>
#include <vector>

#include <pcl/io/pcd_io.h>
#include <pcl/point_types.h>

const float FACTOR = 1000.0f;
const float OFFSET = 10.0f;

struct LidarIntrinsics {
  int width = 2048;
  int height = 128;
  float horizontal_fov = 6.28319;
  float vertical_fov = 0.73584;
  float start_azimuth = 0.0;
  float end_azimuth = 6.28319;
  float start_elevation = 1.19763;
  float end_elevation = 1.93347;
  float rads_per_pixel_azimuth = horizontal_fov * 1.0 / (width - 1);
  float rads_per_pixel_elevation = vertical_fov * 1.0 / (height - 1);
};

std::vector<float> load16BitImage(const std::string& filename,
                                  const float factor, const float offset) {
  int width, height, num_channels;
  uint16_t* image_data =
      stbi_load_16(filename.c_str(), &width, &height, &num_channels, 0);

  std::vector<float> float_image_data(height * width);
  for (int lin_idx = 0; lin_idx < float_image_data.size(); lin_idx++) {
    float_image_data[lin_idx] =
        static_cast<float>(image_data[lin_idx]) / factor - offset;
  }
  stbi_image_free(image_data);
  return float_image_data;
}

__host__ __device__ inline int idivup(int a, int b) {
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

template <typename ElementType>
__host__ __device__ inline ElementType access(int row_idx, int col_idx,
                                              int cols,
                                              const ElementType* data) {
  return data[row_idx * cols + col_idx];
}

__device__ __host__ inline Eigen::Vector3f retrievePoint(
    const float v, const float u, const float* depth_image,
    const float* height_image, const int w, const int h,
    const LidarIntrinsics& lidar_intrinsics) {
  Eigen::Vector3f p = Eigen::Vector3f::Zero();
  float depth = access<float>(v, u, w, depth_image);
  if (depth <= 1e-4) return p;
  float height = access<float>(v, u, w, height_image);
  float r = sqrt(depth * depth - height * height);
  float azimuth_angle_rad = M_PI - u * lidar_intrinsics.rads_per_pixel_azimuth;
  p(0) = r * cos(azimuth_angle_rad);
  p(1) = r * sin(azimuth_angle_rad);
  p(2) = height;
  return p;
}

__global__ void computeNormalImage(float* depth_image, float* height_image,
                                   float* normal_image, const int w,
                                   const int h,
                                   const LidarIntrinsics lidar_intrinsics) {
  // method 1:
  // int tid = blockDim.x * blockIdx.x + threadIdx.x;
  // int stride = blockDim.x;
  // for (int u = tid; u < w; u += stride) {
  //   for (int v = 0; v < h; v++) {
  //     int uu, vv;
  //     if (u == w - 1) uu = 0;
  //     if (v == h - 1) vv = 0;
  //     normal_image_x[v * w + u] =
  //         depth_image[v * w + u] - depth_image[vv * w + uu];
  //   }
  // }

  // method 2:
  // int tid = threadIdx.x;
  // int u_stride = blockDim.x;
  // int v_stride = 128 / 16;
  // for (int u = tid; u < w; u += u_stride) {
  //   for (int v = blockIdx.x * v_stride; v < (blockIdx.x + 1) * v_stride; v++)
  //   {
  //     int uu, vv;
  //     if (u == w - 1) uu = 0;
  //     if (v == h - 1) vv = 0;
  //     normal_image_x[v * w + u] =
  //         depth_image[v * w + u] - depth_image[vv * w + uu];
  //   }
  // }

  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int u_stride = blockDim.x;
  int v_stride = 1;
  for (int u = tid; u < w; u += u_stride) {
    for (int v = 0; v < h; v += v_stride) {
      normal_image[3 * (v * w + u)] = 0.0f;
      normal_image[3 * (v * w + u) + 1] = 0.0f;
      normal_image[3 * (v * w + u) + 2] = 0.0f;

      float sign = 1.0f;
      int uu, vv;
      if (u == w - 1) {
        uu = 0;
      } else {
        uu = u + 1;
      }
      if (v == h - 1) {
        vv = 0;
        sign *= -1.0f;
      } else {
        vv = v + 1;
      }

      float d = access<float>(v, u, w, depth_image);
      float d1 = access<float>(v, uu, w, depth_image);
      float d2 = access<float>(vv, u, w, depth_image);
      if (fabs(d - d1) > 1.0 * d) continue;
      if (fabs(d - d2) > 1.0 * d) continue;

      Eigen::Vector3f p = retrievePoint(v, u, depth_image, height_image, w, h,
                                        lidar_intrinsics);
      Eigen::Vector3f p1 = retrievePoint(v, uu, depth_image, height_image, w, h,
                                         lidar_intrinsics);
      Eigen::Vector3f p2 = retrievePoint(vv, u, depth_image, height_image, w, h,
                                         lidar_intrinsics);
      Eigen::Vector3f n = ((p1 - p).cross(p2 - p)).normalized() * sign;
      normal_image[3 * (v * w + u)] = n.x();
      normal_image[3 * (v * w + u) + 1] = n.y();
      normal_image[3 * (v * w + u) + 2] = n.z();
    }
  }
}

int main(int argc, char** argv) {
  int width = 2048;
  int height = 128;

  // start: read the image
  std::vector<float> depth_image = load16BitImage(
      std::string(
          "/Spy/dataset/mapping_results/nvblox/20220216_garden_day/seq-01/"
          "frame-000000.depth.png"),
      FACTOR, 0.0f);
  std::vector<float> height_image = load16BitImage(
      std::string(
          "/Spy/dataset/mapping_results/nvblox/20220216_garden_day/seq-01/"
          "frame-000000.height.png"),
      FACTOR, OFFSET);

  LidarIntrinsics lidar_intrinsics;
  printf("OSLidar intrinsics--------------------\n");
  printf("width: %d\n", lidar_intrinsics.width);
  printf("height: %d\n", lidar_intrinsics.height);
  printf("horizontal_fov_rad: %f\n", lidar_intrinsics.horizontal_fov);
  printf("vertical_fov_rad: %f\n", lidar_intrinsics.vertical_fov);
  printf("start_elevation: %f\n", lidar_intrinsics.start_elevation);
  printf("end_elevation: %f\n", lidar_intrinsics.end_elevation);
  printf("rads_per_pixel_azimuth: %f\n",
         lidar_intrinsics.rads_per_pixel_azimuth);
  printf("rads_per_pixel_elevation: %f\n",
         lidar_intrinsics.rads_per_pixel_elevation);

  //////////////////////////////// function 1: compute normal images

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, NULL);

  float* depth_image_cuda;
  float* height_image_cuda;
  hipMalloc((void**)&depth_image_cuda, sizeof(float) * 2048 * 128);
  hipMemcpy(depth_image_cuda, depth_image.data(), sizeof(float) * 2048 * 128,
             hipMemcpyHostToDevice);
  hipMalloc((void**)&height_image_cuda, sizeof(float) * 2048 * 128);
  hipMemcpy(height_image_cuda, height_image.data(), sizeof(float) * 2048 * 128,
             hipMemcpyHostToDevice);

  float* normal_image_cuda;
  hipMalloc((void**)&normal_image_cuda, sizeof(float) * 2048 * 128 * 3);

  int block_size = 512;
  int grid_size = 1;
  computeNormalImage<<<grid_size, block_size>>>(
      depth_image_cuda, height_image_cuda, normal_image_cuda, width, height,
      lidar_intrinsics);

  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  float msecTotal = 1.0f;
  hipEventElapsedTime(&msecTotal, start, stop);
  std::cout << "runtime: " << msecTotal << "ms" << std::endl;

  float* normal_image;
  normal_image = (float*)malloc(sizeof(float) * 2048 * 128 * 3);
  hipMemcpy(normal_image, normal_image_cuda, sizeof(float) * 2048 * 128 * 3,
             hipMemcpyDeviceToHost);

  // end: free the memory
  hipFree(depth_image_cuda);
  hipFree(height_image_cuda);
  hipFree(normal_image_cuda);

  //////////////////////////////// converted into PCL
  pcl::PointCloud<pcl::PointXYZINormal> cloud;
  for (int u = 0; u < width; u++) {
    for (int v = 0; v < height; v++) {
      float depth = access<float>(v, u, width, depth_image.data());
      if (depth <= 1e-4) continue;
      float height = access<float>(v, u, width, height_image.data());
      float* normal = new float[3];
      normal[0] = normal_image[3 * (v * width + u)];
      normal[1] = normal_image[3 * (v * width + u) + 1];
      normal[2] = normal_image[3 * (v * width + u) + 2];
      float r = sqrt(depth * depth - height * height);
      float azimuth_angle_rad =
          M_PI - u * lidar_intrinsics.rads_per_pixel_azimuth;
      pcl::PointXYZINormal point;
      point.x = r * cos(azimuth_angle_rad);
      point.y = r * sin(azimuth_angle_rad);
      point.z = height;
      point.normal_x = normal[0];
      point.normal_y = normal[1];
      point.normal_z = normal[2];
      cloud.push_back(point);
    }
  }
  pcl::PCDWriter pcd_writer;
  pcd_writer.write(
      "/Spy/dataset/mapping_results/nvblox/20220216_garden_day/"
      "test_xyz_normal.pcd",
      cloud);

  return 0;
}